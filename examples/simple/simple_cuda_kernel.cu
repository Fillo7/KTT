
#include <hip/hip_runtime.h>
extern "C" __global__ void simpleKernel(const float* a, const float* b, float* result)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    result[index] = a[index] + b[index];
}
