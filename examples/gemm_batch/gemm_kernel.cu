
#include <hip/hip_runtime.h>
#define REAL float

#if GRANULARITY == 1

extern "C" __global__ void gemm_batch(const REAL* A, const REAL* B, REAL* C, int n) {
	int matrix = blockIdx.x*GROUP_SIZE_X + threadIdx.x;
	int tid = threadIdx.x;
/* preload data */
#if CACHING_STRATEGY == 1
    int preloadStartA = blockIdx.x*GROUP_SIZE_X*SIZE_A*SIZE_B;
    int preloadStartB = blockIdx.x*GROUP_SIZE_X*SIZE_C*SIZE_A;
	__shared__ REAL bufA[GROUP_SIZE_X*SIZE_A*SIZE_B];
	__shared__ REAL bufB[GROUP_SIZE_X*SIZE_C*SIZE_A];
	for (int i = 0; i < SIZE_A*SIZE_B; i++)
		bufA[i*GROUP_SIZE_X+tid] = A[preloadStartA + i*GROUP_SIZE_X + tid];
	for (int i = 0; i < SIZE_C*SIZE_A; i++)
        bufB[i*GROUP_SIZE_X+tid] = B[preloadStartB + i*GROUP_SIZE_X + tid];
	__syncthreads();
#elif CACHING_STRATEGY == 2
    int preloadStartA = matrix*SIZE_A*SIZE_B;
    int preloadStartB = matrix*SIZE_C*SIZE_A;
    REAL bufA[SIZE_A*SIZE_B];
    REAL bufB[SIZE_C*SIZE_A];
    for (int i = 0; i < SIZE_A*SIZE_B; i++)
        bufA[i] = A[preloadStartA + i];
    for (int i = 0; i < SIZE_C*SIZE_A; i++)
        bufB[i] = B[preloadStartB + i];
#endif

/* offsets into memory */
#if CACHING_STRATEGY == 0
    int startA = matrix*SIZE_A*SIZE_B;
    int startB = matrix*SIZE_C*SIZE_A;
#else
    int startA = tid*SIZE_A*SIZE_B;
    int startB = tid*SIZE_C*SIZE_A;
#endif
    int startC = matrix*SIZE_C*SIZE_B;

/* compute multiplication */
    for (int i = 0; i < SIZE_B; i++)
        for (int j = 0; j < SIZE_C; j++) {
            REAL tmp = (REAL)0.0;
            for (int k = 0; k < SIZE_A; k++)
#if CACHING_STRATEGY == 0
                tmp += A[startA + i*SIZE_A + k] * B[startB + k*SIZE_C + j];
#endif
#if CACHING_STRATEGY == 1
                tmp += bufA[startA + i*SIZE_A + k] * bufB[startB + k*SIZE_C + j];
#endif
#if CACHING_STRATEGY == 2
                tmp += bufA[i*SIZE_A + k] * bufB[k*SIZE_C + j];
#endif
            C[startC + i*SIZE_C + j] = tmp;
        }
}

#endif

#if GRANULARITY == 2
extern "C" __global__ void gemm_batch(const REAL* A, const REAL* B, REAL* C, int n) {
    int matrix = blockIdx.x*MGCG_GROUP_SIZE_Y + threadIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

/* preload data */
#if CACHING_STRATEGY > 0
    int preloadStartA = blockIdx.x*MGCG_GROUP_SIZE_Y*SIZE_A*SIZE_B;
    int preloadStartB = blockIdx.x*MGCG_GROUP_SIZE_Y*SIZE_C*SIZE_A;
    int myOffset = ty*SIZE_C + tx;
    __shared__ REAL bufA[MGCG_GROUP_SIZE_Y*SIZE_A*SIZE_B];
    #if CACHING_STRATEGY == 1
    __shared__ REAL bufB[MGCG_GROUP_SIZE_Y*SIZE_C*SIZE_A];
    #endif
    for (int i = myOffset; i < SIZE_A*SIZE_B*MGCG_GROUP_SIZE_Y; i+= SIZE_C*MGCG_GROUP_SIZE_Y)
        bufA[i] = A[preloadStartA + i];
    #if CACHING_STRATEGY == 1
     for (int i = myOffset; i < SIZE_C*SIZE_A*MGCG_GROUP_SIZE_Y; i+= SIZE_C*MGCG_GROUP_SIZE_Y)
        bufB[i] = B[preloadStartB + i];
    #endif
    __syncthreads();
#endif
/* offsets into memory */
#if CACHING_STRATEGY == 0
    int startA = matrix*SIZE_A*SIZE_B;
    int startB = matrix*SIZE_C*SIZE_A;
#endif
#if CACHING_STRATEGY == 1
    int startA = ty*SIZE_A*SIZE_B;
    int startB = ty*SIZE_C*SIZE_A;
#endif
#if CACHING_STRATEGY == 2
    int startA = ty*SIZE_A*SIZE_B;
    int startB = matrix*SIZE_C*SIZE_A;
#endif
    int startC = matrix*SIZE_C*SIZE_B;

/* compute multiplication */
#if CACHING_STRATEGY == 2
    REAL tmp[SIZE_B];
    for (int i = 0; i < SIZE_B; i++) 
        tmp[i] = (REAL)0.0;
    for (int i = 0; i < SIZE_A; i++) {
        REAL myB = B[startB + i*SIZE_C + tx];//bufB[startB + i*SIZE_C + tx];
        for (int j = 0; j < SIZE_B; j++)
            tmp[j] += bufA[startA + j*SIZE_A + i] * myB;
    }
    for (int i = 0; i < SIZE_B; i++)
        C[startC + i*SIZE_C + tx] = tmp[i];
#else
    for (int i = 0; i < SIZE_B; i++) {
        REAL tmp = (REAL)0.0;
        for (int k = 0; k < SIZE_A; k++)
    #if CACHING_STRATEGY == 0
            tmp += A[startA + i*SIZE_A + k] * B[startB + k*SIZE_C + tx];
    #endif
    #if CACHING_STRATEGY == 1
            tmp += bufA[startA + i*SIZE_A + k] * bufB[startB + k*SIZE_C + tx];
    #endif
        C[startC + i*SIZE_C + tx] = tmp;
    }
#endif
}
#endif

#if GRANULARITY == 3
extern "C" __global__ void gemm_batch(const REAL* A, const REAL* B, REAL* C, int n) {
    int matrix = blockIdx.x;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

/* preload data */
#if CACHING_STRATEGY == 1
    int preloadStartA = blockIdx.x*SIZE_A*SIZE_B;
    int preloadStartB = blockIdx.x*SIZE_C*SIZE_A;
    int myOffset = ty*SIZE_C + tx;
    __shared__ REAL bufA[SIZE_A*SIZE_B];
    __shared__ REAL bufB[SIZE_C*SIZE_A];
    for (int i = myOffset; i < SIZE_A*SIZE_B; i+= SIZE_C*MGCG_GROUP_SIZE_Y)
        bufA[i] = A[preloadStartA + i];
     for (int i = myOffset; i < SIZE_C*SIZE_A; i+= SIZE_C*MGCG_GROUP_SIZE_Y)
        bufB[i] = B[preloadStartB + i];
    __syncthreads();
#endif
/* offsets into memory */
#if CACHING_STRATEGY == 0
    int startA = matrix*SIZE_A*SIZE_B;
    int startB = matrix*SIZE_C*SIZE_A;
#else
    int startA = 0;
    int startB = 0;
#endif
    int startC = matrix*SIZE_C*SIZE_B;

/* compute multiplication */
    for (int i = ty; i < SIZE_B; i+= MGCG_GROUP_SIZE_Y) {
        REAL tmp = (REAL)0.0;
        for (int k = 0; k < SIZE_A; k++)
#if CACHING_STRATEGY == 0
            tmp += A[startA + i*SIZE_A + k] * B[startB + k*SIZE_C + tx];
#endif
#if CACHING_STRATEGY == 1
            tmp += bufA[startA + i*SIZE_A + k] * bufB[startB + k*SIZE_C + tx];
#endif
        C[startC + i*SIZE_C + tx] = tmp;
    }
}
#endif
