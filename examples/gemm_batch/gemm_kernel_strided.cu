
#include <hip/hip_runtime.h>
#define REAL float

#if GRANULARITY == 1

extern "C" __global__ void gemm_batch(const REAL* A, const REAL* B, REAL* C, int n) {
	int mBlock = blockIdx.x*MGCG_GROUP_SIZE_Y+threadIdx.y;
	int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = ty*STRIDE_BLOCK + tx;
/* preload data */
#if CACHING_STRATEGY == 1
    int preloadStartA = blockIdx.x*MGCG_GROUP_SIZE_Y*STRIDE_BLOCK*SIZE_A*SIZE_B;
    int preloadStartB = blockIdx.x*MGCG_GROUP_SIZE_Y*STRIDE_BLOCK*SIZE_C*SIZE_A;
	__shared__ REAL bufA[MGCG_GROUP_SIZE_Y*STRIDE_BLOCK*SIZE_A*SIZE_B];
	__shared__ REAL bufB[MGCG_GROUP_SIZE_Y*STRIDE_BLOCK*SIZE_C*SIZE_A];
	for (int i = 0; i < SIZE_A*SIZE_B; i++)
		bufA[i*MGCG_GROUP_SIZE_Y*STRIDE_BLOCK+tid] = A[preloadStartA + i*MGCG_GROUP_SIZE_Y*STRIDE_BLOCK + tid];
	for (int i = 0; i < SIZE_C*SIZE_A; i++)
        bufB[i*MGCG_GROUP_SIZE_Y*STRIDE_BLOCK+tid] = B[preloadStartB + i*MGCG_GROUP_SIZE_Y*STRIDE_BLOCK + tid];
	__syncthreads();
#elif CACHING_STRATEGY == 2
    int preloadStartA = mBlock*STRIDE_BLOCK*SIZE_A*SIZE_B + tx;
    int preloadStartB = mBlock*STRIDE_BLOCK*SIZE_C*SIZE_A + tx;
    REAL bufA[SIZE_A*SIZE_B];
    REAL bufB[SIZE_C*SIZE_A];
    for (int i = 0; i < SIZE_A*SIZE_B; i++)
        bufA[i] = A[preloadStartA + i*STRIDE_BLOCK];
    for (int i = 0; i < SIZE_C*SIZE_A; i++)
        bufB[i] = B[preloadStartB + i*STRIDE_BLOCK];
#endif

/* offsets into memory */
#if CACHING_STRATEGY == 0
    int startA = mBlock*STRIDE_BLOCK*SIZE_A*SIZE_B + tx;
    int startB = mBlock*STRIDE_BLOCK*SIZE_C*SIZE_A + tx;
#else
    int startA = ty*STRIDE_BLOCK*SIZE_A*SIZE_B + tx;
    int startB = ty*STRIDE_BLOCK*SIZE_C*SIZE_A + tx;
#endif
    int startC = mBlock*STRIDE_BLOCK*SIZE_C*SIZE_B + tx;

/* compute multiplication */
    for (int i = 0; i < SIZE_B; i++)
        for (int j = 0; j < SIZE_C; j++) {
            REAL tmp = (REAL)0.0;
            for (int k = 0; k < SIZE_A; k++)
#if CACHING_STRATEGY == 0
                tmp += A[startA + (i*SIZE_A+k)*STRIDE_BLOCK] * B[startB + (k*SIZE_C+j)*STRIDE_BLOCK];
#endif
#if CACHING_STRATEGY == 1
                tmp += bufA[startA + (i*SIZE_A+k)*STRIDE_BLOCK] * bufB[startB + (k*SIZE_C+j)*STRIDE_BLOCK];
#endif
#if CACHING_STRATEGY == 2
                tmp += bufA[i*SIZE_A + k] * bufB[k*SIZE_C + j];
#endif
            C[startC + (i*SIZE_C+j)*STRIDE_BLOCK] = tmp;
        }
}

#endif

